#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <windows.h>
#include <conio.h>
#include <stdio.h>
#include <math.h>

#include "simple_kernel.cuh"
#include "main.h"

__global__ void add_kernel(int* to_be_added, int to_add, int boundary) 
{
	const int x_id = threadIdx.x + blockDim.x * blockIdx.x;
	if (x_id >= boundary) {
		return;
	}

	to_be_added[x_id] = to_be_added[x_id] + to_add;
}

void cu_control(int to_add, int boundary)
{
	thrust::host_vector<int> hv_1(5);

	cmake_t t1;
	t1.mem_a = 1;
	t1.mem_b = 2;

	int* to_be_added = new int[boundary];
	memset(to_be_added, 0, boundary * sizeof(int));

	int* device_to_be_added;
	hipMalloc((void**)&device_to_be_added, boundary * sizeof(int));
	hipMemcpy(device_to_be_added, to_be_added, boundary * sizeof(int), hipMemcpyHostToDevice);

	int add_blkSettings = ceil(boundary * 1.0 / 256);
	add_kernel << <add_blkSettings, 256 >> > (device_to_be_added, to_add, boundary);

	hipMemcpy(to_be_added, device_to_be_added, boundary * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < boundary; i++)
		printf("%d\n", to_be_added[i]);
		
	delete[] to_be_added;
}